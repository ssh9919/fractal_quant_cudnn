#include "hip/hip_runtime.h"
/*
   Copyright 2015 Kyuyeon Hwang (kyuyeon.hwang@gmail.com)

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include "CudaKernels.h"
#include <stdio.h>
#define THREAD_PER_BLOCK 512


namespace fractal
{

namespace cudaKernels
{


template<class T>
inline __device__ T _exp(const T x);

template<class T>
inline __device__ T _log(const T x);

template<class T>
inline __device__ T _sqrt(const T x);

template<class T>
static __global__ void MemSetKernel(T *x, const T val, const unsigned long n);

template<class T>
static __global__ void ElemMultKernel(const T *x, const T *y, T *z, const unsigned long n);

template<class T>
static __global__ void AddKernel(const T *x, const T *y, T *z, const unsigned long n);

template<class T>
static __global__ void FuncSigmoidKernel(const T *x, T *y, const unsigned long n, FLOAT delta);

template<class T>
static __global__ void FuncTanhKernel(const T *x, T *y, const unsigned long n,FLOAT delta);

template<class T>
static __global__ void FuncSoftplusKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncRectLinearKernel(const T *x, T *y, const unsigned long n,FLOAT delta, int M,int relu_delta_final_decision);

template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, T *y, const T _min, const T _max, const unsigned long n);

template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv, const T decayRate, const unsigned long n);

template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta, const T learningRate, const T decayRate, const unsigned long n);


template<>
inline __device__ float _exp<float>(const float x)
{
    return min(__expf(x), 1e30);
}


template<>
inline __device__ double _exp<double>(const double x)
{
    return min(exp(x), 1e300);
}


template<>
inline __device__ float _log<float>(const float x)
{
    return __logf(x);
}


template<>
inline __device__ double _log<double>(const double x)
{
    return log(x);
}


template<>
inline __device__ float _sqrt<float>(const float x)
{
    return __fsqrt_rn(x);
}


template<>
inline __device__ double _sqrt<double>(const double x)
{
    return __dsqrt_rn(x);
}



template<class T>
static __global__ void MemSetKernel(T *x, const T val, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    x[idx] = val;
}


template<class T>
static __global__ void ElemMultKernel(const T *x, const T *y, T *z, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    z[idx] = x[idx] * y[idx];
}


template<class T>
static __global__ void AddKernel(const T *x, const T *y, T *z, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    z[idx] = x[idx] + y[idx];
}


template<class T>
static __global__ void FuncSigmoidKernel(const T *x, T *y, const unsigned long n, FLOAT delta)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = (T)1 / ((T)1 + _exp<T>(-x[idx]));
#if QUANT_RELU 
if((delta <101.0 && delta > 99.0) == 0)
{
       //if(threadIdx.x == 1) printf("sig before : %f\n",y[idx]);
    y[idx] = floor((fabs(y[idx])/delta)+(T)0.5);
	y[idx] = y[idx]*delta;
      // if(threadIdx.x == 1) printf("sig after : %f\n",y[idx]);
}
#endif
}


template<class T>
static __global__ void FuncTanhKernel(const T *x, T *y, const unsigned long n, FLOAT delta)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = _exp<T>((T)(-2) * x[idx]);

    y[idx] = (T)2 / ((T)1 + v) - (T)1;
#if QUANT_RELU
if((delta <101.0 && delta > 99.0) == 0)
{
	T sign_;
	sign_ = signbit(y[idx]); 
	
       //if(threadIdx.x == 1) printf("tanh before : %f\n",y[idx]);
	if(sign_ != 0)	
		y[idx] = -1 * min(floor((fabs(y[idx])/delta)+(T)0.5),(1/delta));
	else 
		y[idx] = min(floor((fabs(y[idx])/delta)+(T)0.5),(1/delta));
	
	y[idx] = y[idx]*delta;
       //if(threadIdx.x == 1) printf("tanh after : %f\n",y[idx]);
}
#endif
}


template<class T>
static __global__ void FuncSoftplusKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = _log<T>((T)1 + _exp<T>(x[idx]));
}


template<class T>
static __global__ void FuncRectLinearKernel(const T *x, T *y, const unsigned long n,FLOAT delta, int M,int relu_delta_final_decision)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    //y[idx] = max((T)0, x[idx]);
    /* Leaky */
    y[idx] = max((T)0.01 * x[idx], x[idx]);
#if QUANT_RELU
if(relu_delta_final_decision == 1)
{
	//if(threadIdx.x == 1 )printf("pre : %f\n",y[idx]);
	y[idx] = min((T)floor((fabs(y[idx])/delta)+(T)0.5),(T)M);
	y[idx] = y[idx]*delta;
	//if(threadIdx.x == 1) printf("after : %f\n",y[idx]);
}
	
#endif
}


template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, T *y, const unsigned long n)
{
        __shared__ T _v[THREAD_PER_BLOCK];
        T v_tmp, v_max;
        unsigned long i;

        x += blockIdx.x * n;
        y += blockIdx.x * n;


        /* Sequential reduction(max) */
        v_tmp = threadIdx.x < n ? x[threadIdx.x] : (FLOAT) 0;

        #pragma unroll
        for(i = threadIdx.x + blockDim.x; i < n; i += blockDim.x)
        {
                v_tmp = max(v_tmp, x[i]);
        }

    _v[threadIdx.x] = v_tmp;

        __syncthreads();

        /* Parallel reduction(max) */
        i = (blockDim.x >> 1);
        if(threadIdx.x < i)
                v_tmp = _v[threadIdx.x];

        for(; i > 0; i >>= 1)
        {
                if(threadIdx.x < i && threadIdx.x + i < n)
                {
                        v_tmp = max(v_tmp, _v[threadIdx.x + i]);
                        _v[threadIdx.x] = v_tmp;
                }

                __syncthreads();
        }

    v_max = _v[0];

    __syncthreads();

        /* Sequential reduction(+) */
        v_tmp = (T) 0;

        #pragma unroll
        for(i = threadIdx.x; i < n; i += blockDim.x)
        {
                v_tmp += _exp<T>(x[i] - v_max);
        }

    _v[threadIdx.x] = v_tmp;

        __syncthreads();

        /* Parallel reduction(+) */
        i = (blockDim.x >> 1);
        if(threadIdx.x < i)
                v_tmp = _v[threadIdx.x];

        for(; i > 0; i >>= 1)
        {
                if(threadIdx.x < i)
                {
                        v_tmp += _v[threadIdx.x + i];
                        _v[threadIdx.x] = v_tmp;
                }

                __syncthreads();
        }


    /* Update */
        v_tmp = _v[0];

        #pragma unroll
        for(i = threadIdx.x; i < n; i += blockDim.x)
        {
                y[i] = _exp<T>(x[i] - v_max) / v_tmp;
        }
}


template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, T *y, const T _min, const T _max, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = min(_max, max(_min, x[idx]));
}


template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = x[idx];
    y[idx] = v * ((T)1 - v);
}


template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = x[idx];
    y[idx] = ((T)1 - v) * ((T)1 + v);
}


template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = (T)1 - _exp<T>(-x[idx]);
}


template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    //y[idx] = (T)(x[idx] > (T)0);
    /* Leaky */
    y[idx] = (T)0.01 + (T)0.99 * (T)(x[idx] > (T)0);
}


template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv, const T decayRate, const unsigned long n)
{
    unsigned long idx;
    T ms, rms, deriv;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    ms = msDeriv[idx];
    deriv = derivs[idx];

    T bound = _sqrt<T>((T)1 / ((T)1 - decayRate));

    ms = decayRate * ms + ((T)1 - decayRate) * deriv * deriv;
    rms = _sqrt<T>(ms) + (T)1e-20;

    newDerivs[idx] = min(bound, max(-bound, deriv / rms));
    msDeriv[idx] = ms;
}


template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta, const T learningRate, const T decayRate, const unsigned long n)
{
    unsigned long idx;
    T _msDelta, rmsDelta;
    T _msDeriv, rmsDeriv;
    T deriv, delta;

    const T bound = (T)10;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    _msDeriv = msDeriv[idx];
    _msDelta = msDelta[idx];
    deriv = derivs[idx];

    _msDeriv = decayRate * _msDeriv + ((T)1 - decayRate) * deriv * deriv;
    rmsDeriv = _sqrt<T>(_msDeriv) + (T)1e-20;

    rmsDelta = _sqrt<T>(_msDelta + learningRate * learningRate);

    delta = rmsDelta * min(bound, max(-bound, deriv / rmsDeriv));

    _msDelta = decayRate * _msDelta + ((T)1 - decayRate) * delta * delta;

    deltas[idx] = delta;
    msDeriv[idx] = _msDeriv;
    msDelta[idx] = _msDelta;
}


template<class T>
void MemSet(T *_x, const T val, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MemSetKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, val, n);
}


template<class T>
void ElemMult(const T *_x, const T *_y, T *_z, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    ElemMultKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _z, n);
}


template<class T>
void Add(const T *_x, const T *_y, T *_z, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    AddKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _z, n);
}


template<class T>
void FuncSigmoid(const T *_x, T *_y, const unsigned long n, const hipStream_t stream,FLOAT delta)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n, delta);
}


template<class T>
void FuncTanh(const T *_x, T *_y, const unsigned long n, const hipStream_t stream,FLOAT delta)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n, delta);
}


template<class T>
void FuncSoftplus(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncRectLinear(const T *_x, T *_y, const unsigned long n, const hipStream_t stream, FLOAT delta, int M,int relu_delta_final_decision)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n,delta,M,relu_delta_final_decision);
}


template<class T>
void FuncSoftmax(const T *_x, T *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream)
{
    dim3 dimGrid(batchSize);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftmaxKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, layerSize);
}


template<class T>
void FuncBoundRange(const T *_x, T *_y, const T min, const T max, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncBoundRangeKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, min, max, n);
}


template<class T>
void FuncSigmoidDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncTanhDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncSoftplusDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncRectLinearDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void Rmsprop(T *_newDerivs, const T *_derivs, T *_msDeriv, const T decayRate, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    RmspropKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_newDerivs, _derivs, _msDeriv, decayRate, n);
}


template<class T>
void Adadelta(T *_deltas, const T *_derivs, T *_msDeriv, T *_msDelta, const T learningRate, const T decayRate, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    AdadeltaKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_deltas, _derivs, _msDeriv, _msDelta, learningRate, decayRate, n);
}


template void MemSet<float>(float *_x, const float val, const unsigned long n, const hipStream_t stream);
template void MemSet<double>(double *_x, const double val, const unsigned long n, const hipStream_t stream);

template void ElemMult<float>(const float *_x, const float *_y, float *_z, const unsigned long n, const hipStream_t stream);
template void ElemMult<double>(const double *_x, const double *_y, double *_z, const unsigned long n, const hipStream_t stream);

template void Add<float>(const float *_x, const float *_y, float *_z, const unsigned long n, const hipStream_t stream);
template void Add<double>(const double *_x, const double *_y, double *_z, const unsigned long n, const hipStream_t stream);

template void FuncSigmoid<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream, FLOAT delta);
template void FuncSigmoid<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream, FLOAT delta);

template void FuncTanh<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream,FLOAT delta);
template void FuncTanh<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream,FLOAT delta);

template void FuncSoftplus<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSoftplus<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncRectLinear<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream, FLOAT delta,int M,int relu_delta_final_decision);
template void FuncRectLinear<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream, FLOAT delta, int M,int relu_delta_final_decision);

template void FuncSoftmax<float>(const float *_x, float *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream);
template void FuncSoftmax<double>(const double *_x, double *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream);

template void FuncBoundRange<float>(const float *_x, float *_y, const float min, const float max, const unsigned long n, const hipStream_t stream);
template void FuncBoundRange<double>(const double *_x, double *_y, const double min, const double max, const unsigned long n, const hipStream_t stream);

template void FuncSigmoidDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSigmoidDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncTanhDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncTanhDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncSoftplusDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSoftplusDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncRectLinearDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncRectLinearDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void Rmsprop<float>(float *_newDerivs, const float *_derivs, float *_msDeriv, const float decayRate, const unsigned long n, const hipStream_t stream);
template void Rmsprop<double>(double *_newDerivs, const double *_derivs, double *_msDeriv, const double decayRate, const unsigned long n, const hipStream_t stream);

template void Adadelta<float>(float *_deltas, const float *_derivs, float *_msDeriv, float *_msDelta, const float learningRate, const float decayRate, const unsigned long n, const hipStream_t stream);
template void Adadelta<double>(double *_deltas, const double *_derivs, double *_msDeriv, double *_msDelta, const double learningRate, const double decayRate, const unsigned long n, const hipStream_t stream);

}

}

