#include "hip/hip_runtime.h"
/*
   Copyright 2015 Kyuyeon Hwang (kyuyeon.hwang@gmail.com)

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include "CudaKernels.h"
#include <stdio.h>
#define THREAD_PER_BLOCK 512


namespace fractal
{

namespace cudaKernels
{


template<class T>
inline __device__ T _exp(const T x);

template<class T>
inline __device__ T _log(const T x);

template<class T>
inline __device__ T _sqrt(const T x);

template<class T>
static __global__ void MemSetKernel(T *x, const T val, const unsigned long n);

template<class T>
static __global__ void ElemMultKernel(const T *x, const T *y, T *z, const unsigned long n);

template<class T>
static __global__ void AddKernel(const T *x, const T *y, T *z, const unsigned long n);

template<class T>
static __global__ void FuncSigmoidKernel(const T *x, T *y, const unsigned long n, FLOAT delta);

/* IBM check start */
/* template signal quantization for tanh */
template<class T>
static __global__ void FuncTanhKernel(const T *x, T *y, const unsigned long n,FLOAT delta);
/* template signal quantization for tanh */
/* IBM check end */

/* IBM check start */
/* template weight quantization */
template<class T> 
static __global__ void WeightQuantKernel(const T *x, T *y, const unsigned long n,FLOAT delta, int M);
/* template weight quantization */
/* IBM check end */

template<class T>
static __global__ void FuncSoftplusKernel(const T *x, T *y, const unsigned long n);

/* IBM check start */
/* template signal quantization for rectlinear */
template<class T>
static __global__ void FuncRectLinearKernel(const T *x, T *y,T *y_fixed , const unsigned long n,FLOAT delta, int M,int relu_delta_final_decision);
/* template signal quantization for rectlinear */
/* IBM check end */

template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, T *y, const T _min, const T _max, const unsigned long n);

template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, T *y, const unsigned long n);

template<class T>
    static __global__ void GenerateDropoutMaskKernel(T *mask, const T *uniformDist, const unsigned long n, const T dropoutRate);

template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv, const T decayRate, const unsigned long n);

template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta, const T learningRate, const T decayRate, const unsigned long n);


template<>
inline __device__ float _exp<float>(const float x)
{
    return min(__expf(x), 1e30);
}


template<>
inline __device__ double _exp<double>(const double x)
{
    return min(exp(x), 1e300);
}


template<>
inline __device__ float _log<float>(const float x)
{
    return __logf(x);
}


template<>
inline __device__ double _log<double>(const double x)
{
    return log(x);
}


template<>
inline __device__ float _sqrt<float>(const float x)
{
    return __fsqrt_rn(x);
}


template<>
inline __device__ double _sqrt<double>(const double x)
{
    return __dsqrt_rn(x);
}



template<class T>
static __global__ void MemSetKernel(T *x, const T val, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    x[idx] = val;
}


template<class T>
static __global__ void ElemMultKernel(const T *x, const T *y, T *z, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    z[idx] = x[idx] * y[idx];
}


template<class T>
static __global__ void AddKernel(const T *x, const T *y, T *z, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    z[idx] = x[idx] + y[idx];
}


/* IBM check start */
/* Signal quantization kernel for Sigmoid */
/* If the QUANT_RELU flag is on Quantization Model */
template<class T>
static __global__ void FuncSigmoidKernel(const T *x, T *y,T *y_fixed, const unsigned long n, FLOAT delta)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y_fixed[idx] = (T)1 / ((T)1 + _exp<T>(-x[idx]));
    y[idx] = (T)1 / ((T)1 + _exp<T>(-x[idx]));
#if QUANT_RELU // condition for signal quantization 
if((delta <101.0 && delta > 99.0) == 0)
{
	y[idx] = floor((fabs(y_fixed[idx])/delta)+(T)0.5);
	y[idx] = y[idx]*delta;
}
#endif
}
/* Signal quantization kernel for Sigmoid */
/* IBM check end */


/* IBM check start */
/* Signal quantization kernel for Tanh */
/* If the QUANT_RELU flag is on Quantization Model */
template<class T>
static __global__ void FuncTanhKernel(const T *x, T *y, const unsigned long n, FLOAT delta)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = _exp<T>((T)(-2) * x[idx]);

    y[idx] = (T)2 / ((T)1 + v) - (T)1;
#if QUANT_RELU
if((delta <101.0 && delta > 99.0) == 0)
{
	T sign_;
	sign_ = signbit(y[idx]); 
	
	if(sign_ != 0)	
		y[idx] = -1 * min(floor((fabs(y[idx])/delta)+(T)0.5),(1/delta));
	else 
		y[idx] = min(floor((fabs(y[idx])/delta)+(T)0.5),(1/delta));
	
	y[idx] = y[idx]*delta;
}
#endif
}
/* Signal quantization kernel for Tanh */
/* IBM check end */



/* IBM check start */
/* Weight quantization kernel */
template<class T>
static __global__ void WeightQuantKernel(const T *x, T *y, const unsigned long n, FLOAT delta,int M)
{
    unsigned long idx;
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= n) return;
	int sign_;
	sign_ = signbit(x[idx]); 
	
	if(sign_ != 0)	
		y[idx] = (T)-1 * min((T)floor((fabs(x[idx])/delta)+(T)0.5),(T)(M-1)/2);
	else 
		y[idx] = min((T)floor((fabs(x[idx])/delta)+(T)0.5),(T)((M-1)/2));
	
	y[idx] = y[idx]*delta;
}
/* Weight quantization kernel */
/* IBM check end */

template<class T>
static __global__ void FuncSoftplusKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = _log<T>((T)1 + _exp<T>(x[idx]));
}


/* IBM check start */
/* Signal quantization kernel for Rectlinear */
/* If the QUANT_RELU flag is on Quantization Model */
template<class T>
static __global__ void FuncRectLinearKernel(const T *x, T *y, T *y_fixed, const unsigned long n,FLOAT delta, int M,int relu_delta_final_decision)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    //y[idx] = max((T)0, x[idx]);
    /* Leaky */
    y[idx] = max((T)0.01 * x[idx], x[idx]);
    y_fixed[idx] = max((T)0.01 * x[idx], x[idx]);
#if QUANT_RELU
if(relu_delta_final_decision == 1)
{
	//if(threadIdx.x == 1 )printf("pre : %f\n",y[idx]);
	
	y[idx] =  min((T)floor((y[idx]/delta)+(T)0.5),(T)(M-1));

        y[idx] = y[idx]*delta;
	//if(threadIdx.x == 1) printf("after : %f\n",y[idx]);
}
	
#endif
}
/* Signal quantization kernel for Rectlinear */
/* IBM check end */


template<class T>
static __global__ void FuncSoftmaxKernel(const T *x, T *y, const unsigned long n)
{
        __shared__ T _v[THREAD_PER_BLOCK];
        T v_tmp, v_max;
        unsigned long i;

        x += blockIdx.x * n;
        y += blockIdx.x * n;


        /* Sequential reduction(max) */
        v_tmp = threadIdx.x < n ? x[threadIdx.x] : (FLOAT) 0;

        #pragma unroll
        for(i = threadIdx.x + blockDim.x; i < n; i += blockDim.x)
        {
                v_tmp = max(v_tmp, x[i]);
        }

    _v[threadIdx.x] = v_tmp;

        __syncthreads();

        /* Parallel reduction(max) */
        i = (blockDim.x >> 1);
        if(threadIdx.x < i)
                v_tmp = _v[threadIdx.x];

        for(; i > 0; i >>= 1)
        {
                if(threadIdx.x < i && threadIdx.x + i < n)
                {
                        v_tmp = max(v_tmp, _v[threadIdx.x + i]);
                        _v[threadIdx.x] = v_tmp;
                }

                __syncthreads();
        }

    v_max = _v[0];

    __syncthreads();

        /* Sequential reduction(+) */
        v_tmp = (T) 0;

        #pragma unroll
        for(i = threadIdx.x; i < n; i += blockDim.x)
        {
                v_tmp += _exp<T>(x[i] - v_max);
        }

    _v[threadIdx.x] = v_tmp;

        __syncthreads();

        /* Parallel reduction(+) */
        i = (blockDim.x >> 1);
        if(threadIdx.x < i)
                v_tmp = _v[threadIdx.x];

        for(; i > 0; i >>= 1)
        {
                if(threadIdx.x < i)
                {
                        v_tmp += _v[threadIdx.x + i];
                        _v[threadIdx.x] = v_tmp;
                }

                __syncthreads();
        }


    /* Update */
        v_tmp = _v[0];

        #pragma unroll
        for(i = threadIdx.x; i < n; i += blockDim.x)
        {
                y[i] = _exp<T>(x[i] - v_max) / v_tmp;
        }
}


template<class T>
static __global__ void FuncBoundRangeKernel(const T *x, T *y, const T _min, const T _max, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = min(_max, max(_min, x[idx]));
}


template<class T>
static __global__ void FuncSigmoidDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = x[idx];
    y[idx] = v * ((T)1 - v);
}


template<class T>
static __global__ void FuncTanhDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;
    T v;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    v = x[idx];
    y[idx] = ((T)1 - v) * ((T)1 + v);
}


template<class T>
static __global__ void FuncSoftplusDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    y[idx] = (T)1 - _exp<T>(-x[idx]);
}


template<class T>
static __global__ void FuncRectLinearDerivKernel(const T *x, T *y, const unsigned long n)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    //y[idx] = (T)(x[idx] > (T)0);
    /* Leaky */
    y[idx] = (T)0.01 + (T)0.99 * (T)(x[idx] > (T)0);
}

    template<class T>
static __global__ void GenerateDropoutMaskKernel(T *mask, const T *uniformDist, const unsigned long n, const T dropoutRate)
{
    unsigned long idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    mask[idx] = (T)(uniformDist[idx] >= dropoutRate) / ((T)1 - dropoutRate);
}

template<class T>
static __global__ void RmspropKernel(T *newDerivs, const T *derivs, T *msDeriv, const T decayRate, const unsigned long n)
{
    unsigned long idx;
    T ms, rms, deriv;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    ms = msDeriv[idx];
    deriv = derivs[idx];

    T bound = _sqrt<T>((T)1 / ((T)1 - decayRate));

    ms = decayRate * ms + ((T)1 - decayRate) * deriv * deriv;
    rms = _sqrt<T>(ms) + (T)1e-20;

    newDerivs[idx] = min(bound, max(-bound, deriv / rms));
    msDeriv[idx] = ms;
}


template<class T>
static __global__ void AdadeltaKernel(T *deltas, const T *derivs, T *msDeriv, T *msDelta, const T learningRate, const T decayRate, const unsigned long n)
{
    unsigned long idx;
    T _msDelta, rmsDelta;
    T _msDeriv, rmsDeriv;
    T deriv, delta;

    const T bound = (T)10;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= n) return;

    _msDeriv = msDeriv[idx];
    _msDelta = msDelta[idx];
    deriv = derivs[idx];

    _msDeriv = decayRate * _msDeriv + ((T)1 - decayRate) * deriv * deriv;
    rmsDeriv = _sqrt<T>(_msDeriv) + (T)1e-20;

    rmsDelta = _sqrt<T>(_msDelta + learningRate * learningRate);

    delta = rmsDelta * min(bound, max(-bound, deriv / rmsDeriv));

    _msDelta = decayRate * _msDelta + ((T)1 - decayRate) * delta * delta;

    deltas[idx] = delta;
    msDeriv[idx] = _msDeriv;
    msDelta[idx] = _msDelta;
}


template<class T>
void MemSet(T *_x, const T val, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    MemSetKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, val, n);
}


template<class T>
void ElemMult(const T *_x, const T *_y, T *_z, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    ElemMultKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _z, n);
}


template<class T>
void Add(const T *_x, const T *_y, T *_z, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    AddKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _z, n);
}


/* IBM check start */
/* Signal quantization kernel call for Sigmoid */
template<class T>
void FuncSigmoid(const T *_x, T *_y, T *_y_fixed, const unsigned long n, const hipStream_t stream,FLOAT delta)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _y_fixed, n, delta);
}
/* Signal quantization kernel for Sigmoid */
/* IBM check end */

/* IBM check start */
/* Signal quantization kernel call for Tanh */
template<class T>
void FuncTanh(const T *_x, T *_y, const unsigned long n, const hipStream_t stream,FLOAT delta)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n, delta);
}
/* Signal quantization kernel for Tanh */
/* IBM check end */

/* IBM check start */
/* Weight quantization kernel call */
template<class T>
void WeightQuant(const T *_x, T *_y, const unsigned long n,const hipStream_t stream, FLOAT delta, int M)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    WeightQuantKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n, delta, M);
}
/* Weight quantization kernel call */
/* IBM check end */

template<class T>
void FuncSoftplus(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


/* IBM check start */
/* Signal quantization kernel call for Rectlinear*/
template<class T>
void FuncRectLinear(const T *_x, T *_y, T *_y_fixed,const unsigned long n, const hipStream_t stream, FLOAT delta, int M,int relu_delta_final_decision)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, _y_fixed,n,delta,M,relu_delta_final_decision);
}
/* Signal quantization kernel call for rectlinear*/
/* IBM check end */


template<class T>
void FuncSoftmax(const T *_x, T *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream)
{
    dim3 dimGrid(batchSize);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftmaxKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, layerSize);
}


template<class T>
void FuncBoundRange(const T *_x, T *_y, const T min, const T max, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncBoundRangeKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, min, max, n);
}


template<class T>
void FuncSigmoidDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSigmoidDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncTanhDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncTanhDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncSoftplusDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncSoftplusDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}


template<class T>
void FuncRectLinearDeriv(const T *_x, T *_y, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    FuncRectLinearDerivKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_x, _y, n);
}

    template<class T>
void GenerateDropoutMask(T *_mask, const T *_uniformDist, const unsigned long n, const T dropoutRate, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    GenerateDropoutMaskKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_mask, _uniformDist, n, dropoutRate);
}


template<class T>
void Rmsprop(T *_newDerivs, const T *_derivs, T *_msDeriv, const T decayRate, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    RmspropKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_newDerivs, _derivs, _msDeriv, decayRate, n);
}


template<class T>
void Adadelta(T *_deltas, const T *_derivs, T *_msDeriv, T *_msDelta, const T learningRate, const T decayRate, const unsigned long n, const hipStream_t stream)
{
    dim3 dimGrid((n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK);
    dim3 dimBlock(THREAD_PER_BLOCK);

    AdadeltaKernel<T><<<dimGrid, dimBlock, 0, stream>>>(_deltas, _derivs, _msDeriv, _msDelta, learningRate, decayRate, n);
}


template void MemSet<float>(float *_x, const float val, const unsigned long n, const hipStream_t stream);
template void MemSet<double>(double *_x, const double val, const unsigned long n, const hipStream_t stream);

template void ElemMult<float>(const float *_x, const float *_y, float *_z, const unsigned long n, const hipStream_t stream);
template void ElemMult<double>(const double *_x, const double *_y, double *_z, const unsigned long n, const hipStream_t stream);

template void Add<float>(const float *_x, const float *_y, float *_z, const unsigned long n, const hipStream_t stream);
template void Add<double>(const double *_x, const double *_y, double *_z, const unsigned long n, const hipStream_t stream);

/* IBM check start */
template void FuncSigmoid<float>(const float *_x, float *_y, float *_y_fixed,const unsigned long n, const hipStream_t stream, FLOAT delta);
template void FuncSigmoid<double>(const double *_x, double *_y, double *_y_fixed, const unsigned long n, const hipStream_t stream, FLOAT delta);

template void FuncTanh<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream,FLOAT delta);
template void FuncTanh<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream,FLOAT delta);

template void WeightQuant<float>(const float *_x, float *_y, const unsigned long n,const hipStream_t stream, FLOAT delta, int M);
template void WeightQuant<double>(const double *_x, double *_y, const unsigned long n,const hipStream_t stream, FLOAT delta, int M);

template void FuncRectLinear<float>(const float *_x, float *_y, float *_y_fixed,const unsigned long n, const hipStream_t stream, FLOAT delta,int M,int relu_delta_final_decision);
template void FuncRectLinear<double>(const double *_x, double *_y, double *_y_fixed, const unsigned long n, const hipStream_t stream, FLOAT delta, int M,int relu_delta_final_decision);
/* IBM check end */

template void FuncSoftplus<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSoftplus<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);


template void FuncSoftmax<float>(const float *_x, float *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream);
template void FuncSoftmax<double>(const double *_x, double *_y, const unsigned long layerSize, const unsigned long batchSize, const hipStream_t stream);

template void FuncBoundRange<float>(const float *_x, float *_y, const float min, const float max, const unsigned long n, const hipStream_t stream);
template void FuncBoundRange<double>(const double *_x, double *_y, const double min, const double max, const unsigned long n, const hipStream_t stream);

template void FuncSigmoidDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSigmoidDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncTanhDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncTanhDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncSoftplusDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncSoftplusDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void FuncRectLinearDeriv<float>(const float *_x, float *_y, const unsigned long n, const hipStream_t stream);
template void FuncRectLinearDeriv<double>(const double *_x, double *_y, const unsigned long n, const hipStream_t stream);

template void GenerateDropoutMask<float>(float *_mask, const float *_uniformDist, const unsigned long n, const float dropoutRate, const hipStream_t stream);
template void GenerateDropoutMask<double>(double *_mask, const double *_uniformDist, const unsigned long n, const double dropoutRate, const hipStream_t stream);

template void Rmsprop<float>(float *_newDerivs, const float *_derivs, float *_msDeriv, const float decayRate, const unsigned long n, const hipStream_t stream);
template void Rmsprop<double>(double *_newDerivs, const double *_derivs, double *_msDeriv, const double decayRate, const unsigned long n, const hipStream_t stream);

template void Adadelta<float>(float *_deltas, const float *_derivs, float *_msDeriv, float *_msDelta, const float learningRate, const float decayRate, const unsigned long n, const hipStream_t stream);
template void Adadelta<double>(double *_deltas, const double *_derivs, double *_msDeriv, double *_msDelta, const double learningRate, const double decayRate, const unsigned long n, const hipStream_t stream);

}

}

